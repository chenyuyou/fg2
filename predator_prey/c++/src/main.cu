#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <random>

#include "flamegpu/flamegpu.h"

#define PRED_PREY_INTERACTION_RADIUS 0.1f
#define SAME_SPECIES_AVOIDANCE_RADIUS 0.035f
#define DELTA_TIME 0.001f
#define PRED_SPEED_ADVANTAGE 2.0f
#define PRED_KILL_DISTANCE 0.02f
#define BOUNDS_WIDTH 2.0f
#define MIN_POSITION -1.0f
#define MAX_POSITION 1.0f
#define PREY_GROUP_COHESION_RADIUS 0.2f
#define GRASS_EAT_DISTANCE 0.02f
#define GRASS_REGROW_CYCLES 100
#define GAIN_FROM_FOOD_PREY 75



typedef struct CSVRow {
    int preyPop;
    int predatorPop;
    int grassPop;
} CSVRow;

std::vector<CSVRow> csvData;

FLAMEGPU_STEP_FUNCTION(recordPopulation) {
    CSVRow row;
    row.predatorPop = FLAMEGPU->agent("predator").count();
    row.preyPop = FLAMEGPU->agent("prey").count();
    row.grassPop = FLAMEGPU->agent("grass").count<int>("available", 1);
    csvData.push_back(row);
}

FLAMEGPU_EXIT_FUNCTION(savePopulationData) {
    std::ofstream outputFile;
    outputFile.open("iterations/PreyPred_Count.csv");
    if (outputFile.is_open()) {
        for (const CSVRow& csvRow : csvData) {
            outputFile << "Prey, " << csvRow.preyPop << ", Predator," << csvRow.predatorPop << ", Grass," << csvRow.grassPop << std::endl;
        }
    }
    else {
        std::cout << "Failed to open file for saving population data!";
    }
    std::cout << "Data saved" << std::endl;
}

CSVRow loadPopulations() {
    std::ifstream inputFile("iterations/initial_populations.txt");
    CSVRow initialPopulations;
    initialPopulations.preyPop = 800;
    initialPopulations.predatorPop = 400;
    initialPopulations.grassPop = 0;
    if (inputFile.is_open()) {
        inputFile >> initialPopulations.preyPop >> initialPopulations.predatorPop >> initialPopulations.grassPop;
    }
    else {
        std::cout << "Warning: Failed to open initial_populations.txt, using default population values" << std::endl;
    }
    return initialPopulations;
}

/*
   The following section of code defines the agent function behaviours in the following format:

   FLAMEGPU_AGENT_FUNCTION(function_name, input_message_type, output_message_type) {
       behaviour definition goes here
   }

*/

// Predator functions
FLAMEGPU_AGENT_FUNCTION(pred_output_location, flamegpu::MessageNone, flamegpu::MessageBruteForce) {
    const flamegpu::id_t id = FLAMEGPU->getID();
    const float x = FLAMEGPU->getVariable<float>("x");
    const float y = FLAMEGPU->getVariable<float>("y");
    FLAMEGPU->message_out.setVariable<flamegpu::id_t>("id", id);
    FLAMEGPU->message_out.setVariable<float>("x", x);
    FLAMEGPU->message_out.setVariable<float>("y", y);

    return flamegpu::ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(pred_follow_prey, flamegpu::MessageBruteForce, flamegpu::MessageNone) {
    // Fetch the predator's position
    const float predator_x = FLAMEGPU->getVariable<float>("x");
    const float predator_y = FLAMEGPU->getVariable<float>("y");

    // Find the closest prey by iterating the prey_location messages
    float closest_prey_x = 0.0f;
    float closest_prey_y = 0.0f;
    float closest_prey_distance = PRED_PREY_INTERACTION_RADIUS;
    int is_a_prey_in_range = 0;

    for (const auto& msg : FLAMEGPU->message_in) {
        // Fetch prey location
        const float prey_x = msg.getVariable<float>("x");
        const float prey_y = msg.getVariable<float>("y");

        // Check if prey is within sight range of predator
        const float dx = predator_x - prey_x;
        const float dy = predator_y - prey_y;
        const float separation = sqrt(dx * dx + dy * dy);

        if (separation < closest_prey_distance) {
            closest_prey_x = prey_x;
            closest_prey_y = prey_y;
            closest_prey_distance = separation;
            is_a_prey_in_range = 1;
        }
    }

    // If there was a prey in range, steer the predator towards it
    if (is_a_prey_in_range) {
        const float steer_x = closest_prey_x - predator_x;
        const float steer_y = closest_prey_y - predator_y;
        FLAMEGPU->setVariable<float>("steer_x", steer_x);
        FLAMEGPU->setVariable<float>("steer_y", steer_y);
    }

    return flamegpu::ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(pred_avoid, flamegpu::MessageBruteForce, flamegpu::MessageNone) {
    // Fetch this predator's position
    const float predator_x = FLAMEGPU->getVariable<float>("x");
    const float predator_y = FLAMEGPU->getVariable<float>("y");
    float avoid_velocity_x = 0.0f;
    float avoid_velocity_y = 0.0f;

    // Add a steering factor away from each other predator. Strength increases with closeness.
    for (const auto& msg : FLAMEGPU->message_in) {
        // Fetch location of other predator
        const float other_predator_x = msg.getVariable<float>("x");
        const float other_predator_y = msg.getVariable<float>("y");

        // Check if the two predators are within interaction radius
        const float dx = predator_x - other_predator_x;
        const float dy = predator_y - other_predator_y;
        const float separation = sqrt(dx * dx + dy * dy);

        if (separation < SAME_SPECIES_AVOIDANCE_RADIUS && separation > 0.0f) {
            avoid_velocity_x += SAME_SPECIES_AVOIDANCE_RADIUS / separation * dx;
            avoid_velocity_y += SAME_SPECIES_AVOIDANCE_RADIUS / separation * dy;
        }
    }

    float steer_x = FLAMEGPU->getVariable<float>("steer_x");
    float steer_y = FLAMEGPU->getVariable<float>("steer_y");
    steer_x += avoid_velocity_x;
    steer_y += avoid_velocity_y;
    FLAMEGPU->setVariable<float>("steer_x", steer_x);
    FLAMEGPU->setVariable<float>("steer_y", steer_y);

    return flamegpu::ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(pred_move, flamegpu::MessageNone, flamegpu::MessageNone) {
    float predator_x = FLAMEGPU->getVariable<float>("x");
    float predator_y = FLAMEGPU->getVariable<float>("y");
    float predator_vx = FLAMEGPU->getVariable<float>("vx");
    float predator_vy = FLAMEGPU->getVariable<float>("vy");
    const float predator_steer_x = FLAMEGPU->getVariable<float>("steer_x");
    const float predator_steer_y = FLAMEGPU->getVariable<float>("steer_y");
    const float predator_life = FLAMEGPU->getVariable<int>("life");

    // Integrate steering forces and cap velocity
    predator_vx += predator_steer_x;
    predator_vy += predator_steer_y;

    float speed = sqrt(predator_vx * predator_vx + predator_vy * predator_vy);
    if (speed > 1.0f) {
        predator_vx /= speed;
        predator_vy /= speed;
    }

    // Integrate velocity
    predator_x += predator_vx * DELTA_TIME * PRED_SPEED_ADVANTAGE;
    predator_y += predator_vy * DELTA_TIME * PRED_SPEED_ADVANTAGE;

    // Bound the position within the environment 
    predator_x = predator_x < MIN_POSITION ? MIN_POSITION : predator_x;
    predator_x = predator_x > MAX_POSITION ? MAX_POSITION : predator_x;
    predator_y = predator_y < MIN_POSITION ? MIN_POSITION : predator_y;
    predator_y = predator_y > MAX_POSITION ? MAX_POSITION : predator_y;

    // Update agent state
    FLAMEGPU->setVariable<float>("x", predator_x);
    FLAMEGPU->setVariable<float>("y", predator_y);
    FLAMEGPU->setVariable<float>("vx", predator_vx);
    FLAMEGPU->setVariable<float>("vy", predator_vy);

    // Reduce life by one unit of energy
    FLAMEGPU->setVariable<int>("life", predator_life - 1);

    return flamegpu::ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(pred_eat_or_starve, flamegpu::MessageBruteForce, flamegpu::MessageNone) {
    const int predator_id = FLAMEGPU->getID();
    int predator_life = FLAMEGPU->getVariable<int>("life");
    int isDead = 0;

    // Iterate prey_eaten messages to see if this predator ate a prey
    for (const auto& msg : FLAMEGPU->message_in) {
        if (msg.getVariable<flamegpu::id_t>("pred_id") == predator_id) {
            predator_life += FLAMEGPU->environment.getProperty<int>("GAIN_FROM_FOOD_PREDATOR");
        }
    }

    // Update agent state
    FLAMEGPU->setVariable<int>("life", predator_life);

    // Did the predator starve?
    if (predator_life < 1) {
        isDead = 1;
    }

    return isDead ? flamegpu::DEAD : flamegpu::ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(pred_reproduction, flamegpu::MessageNone, flamegpu::MessageNone) {
    float random = FLAMEGPU->random.uniform<float>();
    const int currentLife = FLAMEGPU->getVariable<int>("life");
    if (random < FLAMEGPU->environment.getProperty<float>("REPRODUCE_PREDATOR_PROB")) {
        float x = FLAMEGPU->random.uniform<float>() * BOUNDS_WIDTH - BOUNDS_WIDTH / 2.0f;
        float y = FLAMEGPU->random.uniform<float>() * BOUNDS_WIDTH - BOUNDS_WIDTH / 2.0f;
        float vx = FLAMEGPU->random.uniform<float>() * 2 - 1;
        float vy = FLAMEGPU->random.uniform<float>() * 2 - 1;

        FLAMEGPU->setVariable<int>("life", currentLife / 2);
        
        FLAMEGPU->agent_out.setVariable<float>("x", x);
        FLAMEGPU->agent_out.setVariable<float>("y", y);
        FLAMEGPU->agent_out.setVariable<float>("type", 0.0f);
        FLAMEGPU->agent_out.setVariable<float>("vx", vx);
        FLAMEGPU->agent_out.setVariable<float>("vy", vy);
        FLAMEGPU->agent_out.setVariable<float>("steer_x", 0.0f);
        FLAMEGPU->agent_out.setVariable<float>("steer_y", 0.0f);
        FLAMEGPU->agent_out.setVariable<int>("life", currentLife / 2);
    }
    return flamegpu::ALIVE;
}

// Prey functions

FLAMEGPU_AGENT_FUNCTION(prey_output_location, flamegpu::MessageNone, flamegpu::MessageBruteForce) {
    const flamegpu::id_t id = FLAMEGPU->getID();
    const float x = FLAMEGPU->getVariable<float>("x");
    const float y = FLAMEGPU->getVariable<float>("y");
    FLAMEGPU->message_out.setVariable<flamegpu::id_t>("id", id);
    FLAMEGPU->message_out.setVariable<float>("x", x);
    FLAMEGPU->message_out.setVariable<float>("y", y);
    return flamegpu::ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(prey_avoid_pred, flamegpu::MessageBruteForce, flamegpu::MessageNone) {
    // Fetch this prey's position
    const float prey_x = FLAMEGPU->getVariable<float>("x");
    const float prey_y = FLAMEGPU->getVariable<float>("y");
    float avoid_velocity_x = 0.0f;
    float avoid_velocity_y = 0.0f;

    // Add a steering factor away from each predator. Strength increases with closeness.
    for (const auto& msg : FLAMEGPU->message_in) {
        // Fetch location of predator
        const float predator_x = msg.getVariable<float>("x");
        const float predator_y = msg.getVariable<float>("y");

        // Check if the two predators are within interaction radius
        const float dx = prey_x - predator_x;
        const float dy = prey_y - predator_y;
        const float distance = sqrt(dx * dx + dy * dy);

        if (distance < PRED_PREY_INTERACTION_RADIUS) {
            // Steer the prey away from the predator
            avoid_velocity_x += (PRED_PREY_INTERACTION_RADIUS / distance) * dx;
            avoid_velocity_y += (PRED_PREY_INTERACTION_RADIUS / distance) * dy;
        }
    }

    // Update agent state 
    FLAMEGPU->setVariable<float>("steer_x", avoid_velocity_x);
    FLAMEGPU->setVariable<float>("steer_y", avoid_velocity_y);

    return flamegpu::ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(prey_flock, flamegpu::MessageBruteForce, flamegpu::MessageNone) {
    const int   prey_id = FLAMEGPU->getID();
    const float prey_x = FLAMEGPU->getVariable<float>("x");
    const float prey_y = FLAMEGPU->getVariable<float>("y");

    float group_centre_x = 0.0f;
    float group_centre_y = 0.0f;
    float group_velocity_x = 0.0f;
    float group_velocity_y = 0.0f;
    float avoid_velocity_x = 0.0f;
    float avoid_velocity_y = 0.0f;
    int group_centre_count = 0;

    for (const auto& msg : FLAMEGPU->message_in) {
        const int   other_prey_id = msg.getVariable<flamegpu::id_t>("id");
        const float other_prey_x = msg.getVariable<float>("x");
        const float other_prey_y = msg.getVariable<float>("y");
        const float dx = prey_x - other_prey_x;
        const float dy = prey_y - other_prey_y;
        const float separation = sqrt(dx * dx + dy * dy);

        if (separation < PREY_GROUP_COHESION_RADIUS && prey_id != other_prey_id) {
            group_centre_x += other_prey_x;
            group_centre_y += other_prey_y;
            group_centre_count += 1;

            // Avoidance behaviour
            if (separation < SAME_SPECIES_AVOIDANCE_RADIUS) {
                // Was a check for separation > 0 in original - redundant?
                avoid_velocity_x += SAME_SPECIES_AVOIDANCE_RADIUS / separation * dx;
                avoid_velocity_y += SAME_SPECIES_AVOIDANCE_RADIUS / separation * dy;
            }
        }
    }

    // Compute group centre as the average of the nearby prey positions and a velocity to move towards the group centre
    if (group_centre_count > 0) {
        group_centre_x /= group_centre_count;
        group_centre_y /= group_centre_count;
        group_velocity_x = group_centre_x - prey_x;
        group_velocity_y = group_centre_y - prey_y;
    }

    float prey_steer_x = FLAMEGPU->getVariable<float>("steer_x");
    float prey_steer_y = FLAMEGPU->getVariable<float>("steer_y");
    prey_steer_x += group_velocity_x + avoid_velocity_x;
    prey_steer_y += group_velocity_y + avoid_velocity_y;
    FLAMEGPU->setVariable<float>("steer_x", prey_steer_x);
    FLAMEGPU->setVariable<float>("steer_y", prey_steer_y);

    return flamegpu::ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(prey_move, flamegpu::MessageNone, flamegpu::MessageNone) {
    float prey_x = FLAMEGPU->getVariable<float>("x");
    float prey_y = FLAMEGPU->getVariable<float>("y");
    float prey_vx = FLAMEGPU->getVariable<float>("vx");
    float prey_vy = FLAMEGPU->getVariable<float>("vy");
    const float prey_steer_x = FLAMEGPU->getVariable<float>("steer_x");
    const float prey_steer_y = FLAMEGPU->getVariable<float>("steer_y");
    const float prey_life = FLAMEGPU->getVariable<int>("life");

    // Integrate steering forces and cap velocity
    prey_vx += prey_steer_x;
    prey_vy += prey_steer_y;

    float speed = sqrt(prey_vx * prey_vx + prey_vy * prey_vy);
    if (speed > 1.0f) {
        prey_vx /= speed;
        prey_vy /= speed;
    }

    // Integrate velocity
    prey_x += prey_vx * DELTA_TIME;
    prey_y += prey_vy * DELTA_TIME;

    // Bound the position within the environment - can this be moved
    prey_x = prey_x < MIN_POSITION ? MIN_POSITION : prey_x;
    prey_x = prey_x > MAX_POSITION ? MAX_POSITION : prey_x;
    prey_y = prey_y < MIN_POSITION ? MIN_POSITION : prey_y;
    prey_y = prey_y > MAX_POSITION ? MAX_POSITION : prey_y;


    // Update agent state
    FLAMEGPU->setVariable<float>("x", prey_x);
    FLAMEGPU->setVariable<float>("y", prey_y);
    FLAMEGPU->setVariable<float>("vx", prey_vx);
    FLAMEGPU->setVariable<float>("vy", prey_vy);

    // Reduce life by one unit of energy
    FLAMEGPU->setVariable<int>("life", prey_life - 1);

    return flamegpu::ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(prey_eaten, flamegpu::MessageBruteForce, flamegpu::MessageBruteForce) {
    int eaten = 0;
    flamegpu::id_t predator_id = flamegpu::ID_NOT_SET;
    float closest_pred = PRED_KILL_DISTANCE;
    const float prey_x = FLAMEGPU->getVariable<float>("x");
    const float prey_y = FLAMEGPU->getVariable<float>("y");

    // Iterate predator_location messages to find the closest predator
    for (const auto& msg : FLAMEGPU->message_in) {
        // Fetch location of predator
        const float predator_x = msg.getVariable<float>("x");
        const float predator_y = msg.getVariable<float>("y");

        // Check if the two predators are within interaction radius
        const float dx = prey_x - predator_x;
        const float dy = prey_y - predator_y;
        const float distance = sqrt(dx * dx + dy * dy);

        if (distance < closest_pred) {
            predator_id = msg.getVariable<flamegpu::id_t>("id");
            closest_pred = distance;
            eaten = 1;
        }
    }

    if (eaten) {
        FLAMEGPU->message_out.setVariable<flamegpu::id_t>("id", FLAMEGPU->getID());
        FLAMEGPU->message_out.setVariable<flamegpu::id_t>("pred_id", predator_id);
    }

    return eaten ? flamegpu::DEAD : flamegpu::ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(prey_eat_or_starve, flamegpu::MessageBruteForce, flamegpu::MessageNone) {
    // Exercise 3.3

    return flamegpu::ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(prey_reproduction, flamegpu::MessageNone, flamegpu::MessageNone) {
    float random = FLAMEGPU->random.uniform<float>();
    const int currentLife = FLAMEGPU->getVariable<int>("life");
    if (random < FLAMEGPU->environment.getProperty<float>("REPRODUCE_PREY_PROB")) {
        float x = FLAMEGPU->random.uniform<float>() * BOUNDS_WIDTH - BOUNDS_WIDTH / 2.0f;
        float y = FLAMEGPU->random.uniform<float>() * BOUNDS_WIDTH - BOUNDS_WIDTH / 2.0f;
        float vx = FLAMEGPU->random.uniform<float>() * 2 - 1;
        float vy = FLAMEGPU->random.uniform<float>() * 2 - 1;

        FLAMEGPU->setVariable<int>("life", currentLife / 2);
        
        FLAMEGPU->agent_out.setVariable<float>("x", x);
        FLAMEGPU->agent_out.setVariable<float>("y", y);
        FLAMEGPU->agent_out.setVariable<float>("type", 1.0f);
        FLAMEGPU->agent_out.setVariable<float>("vx", vx);
        FLAMEGPU->agent_out.setVariable<float>("vy", vy);
        FLAMEGPU->agent_out.setVariable<float>("steer_x", 0.0f);
        FLAMEGPU->agent_out.setVariable<float>("steer_y", 0.0f);
        FLAMEGPU->agent_out.setVariable<int>("life", currentLife / 2);

    }
    return flamegpu::ALIVE;
}

// Grass functions
FLAMEGPU_AGENT_FUNCTION(grass_output_location, flamegpu::MessageNone, flamegpu::MessageBruteForce) {
    // Exercise 3.1 : Set the variables for the grass_location message
    return flamegpu::ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(grass_eaten, flamegpu::MessageBruteForce, flamegpu::MessageBruteForce) {
    // Exercise 3.2

    return flamegpu::ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(grass_growth, flamegpu::MessageNone, flamegpu::MessageNone) {
    // Exercise 3.4 
    return flamegpu::ALIVE;
}


// Model definition


int main(int argc, const char** argv) {
    NVTX_RANGE("main");
    NVTX_PUSH("ModelDescription");
    flamegpu::ModelDescription model("Tutorial_PredatorPrey_Example");

    /**
     * MESSAGE DEFINITIONS
     */

    {   // Grass location message
        flamegpu::MessageBruteForce::Description& message = model.newMessage("grass_location_message");
        message.newVariable<flamegpu::id_t>("id");
        message.newVariable<float>("x");
        message.newVariable<float>("y");
    }
    {   // Predator location message
        flamegpu::MessageBruteForce::Description& message = model.newMessage("predator_location_message");
        message.newVariable<flamegpu::id_t>("id");
        message.newVariable<float>("x");
        message.newVariable<float>("y");
    }
    {   // Prey location message
        flamegpu::MessageBruteForce::Description& message = model.newMessage("prey_location_message");
        message.newVariable<flamegpu::id_t>("id");
        message.newVariable<float>("x");
        message.newVariable<float>("y");
    }
    {   // Grass eaten message
        flamegpu::MessageBruteForce::Description& message = model.newMessage("grass_eaten_message");
        message.newVariable<flamegpu::id_t>("id");
        message.newVariable<flamegpu::id_t>("prey_id");
    }
    {   // Prey eaten message
        flamegpu::MessageBruteForce::Description& message = model.newMessage("prey_eaten_message");
        message.newVariable<flamegpu::id_t>("id");
        message.newVariable<flamegpu::id_t>("pred_id");
    }


    /**
     * AGENT DEFINITIONS
     */

    {   // Prey agent
        flamegpu::AgentDescription& agent = model.newAgent("prey");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<float>("vx");
        agent.newVariable<float>("vy");
        agent.newVariable<float>("steer_x");
        agent.newVariable<float>("steer_y");
        agent.newVariable<int>("life");
        agent.newVariable<float>("type")
            ;
        auto& fn = agent.newFunction("prey_output_location", prey_output_location);
        fn.setMessageOutput("prey_location_message");
        agent.newFunction("prey_avoid_pred", prey_avoid_pred).setMessageInput("predator_location_message");
        agent.newFunction("prey_flock", prey_flock).setMessageInput("prey_location_message");
        agent.newFunction("prey_move", prey_move);
        auto& function = agent.newFunction("prey_eaten", prey_eaten);
        function.setMessageInput("predator_location_message");
        function.setMessageOutput("prey_eaten_message");
        function.setMessageOutputOptional(true);
        function.setAllowAgentDeath(true);
        auto& fn_prey_eat_or_starve = agent.newFunction("prey_eat_or_starve", prey_eat_or_starve);
        fn_prey_eat_or_starve.setMessageInput("grass_eaten_message");
        fn_prey_eat_or_starve.setAllowAgentDeath(true);
        auto& fn_prey_reproduction = agent.newFunction("prey_reproduction", prey_reproduction);
        fn_prey_reproduction.setAgentOutput("prey", "default");
    }

    {   // Predator agent
        flamegpu::AgentDescription& agent = model.newAgent("predator");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<float>("vx");
        agent.newVariable<float>("vy");
        agent.newVariable<float>("steer_x");
        agent.newVariable<float>("steer_y");
        agent.newVariable<int>("life");
        agent.newVariable<float>("type");

        agent.newFunction("pred_output_location", pred_output_location).setMessageOutput("predator_location_message");
        agent.newFunction("pred_follow_prey", pred_follow_prey).setMessageInput("prey_location_message");
        agent.newFunction("pred_avoid", pred_avoid).setMessageInput("predator_location_message");
        agent.newFunction("pred_move", pred_move);
        auto& fn_pred_eat_or_starve = agent.newFunction("pred_eat_or_starve", pred_eat_or_starve);
        fn_pred_eat_or_starve.setMessageInput("prey_eaten_message");
        fn_pred_eat_or_starve.setAllowAgentDeath(true);
        auto& fn_pred_reproduction = agent.newFunction("pred_reproduction", pred_reproduction);
        fn_pred_reproduction.setAgentOutput("predator", "default");
    }

    {   // Grass agent
        flamegpu::AgentDescription& agent = model.newAgent("grass");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<int>("dead_cycles");
        agent.newVariable<int>("available");
        agent.newVariable<float>("type");
        auto& fn = agent.newFunction("grass_output_location", grass_output_location);
        fn.setMessageOutput("grass_location_message");
        fn.setMessageOutputOptional(true);
        auto& fn_grass_eaten = agent.newFunction("grass_eaten", grass_eaten);
        fn_grass_eaten.setMessageInput("prey_location_message");
        fn_grass_eaten.setMessageOutput("grass_eaten_message");
        fn_grass_eaten.setMessageOutputOptional(true);
        fn_grass_eaten.setAllowAgentDeath(true);
        agent.newFunction("grass_growth", grass_growth);

    }

    /**
      * ENVIRONMENT VARIABLES
      */

    flamegpu::EnvironmentDescription& env = model.Environment();
    env.newProperty<float>("REPRODUCE_PREY_PROB", 0.05f);
    env.newProperty<float>("REPRODUCE_PREDATOR_PROB", 0.03f);
    env.newProperty<int>("GAIN_FROM_FOOD_PREDATOR", 50);

    /**
     * Control flow
     */
    {   // Layer #1
        flamegpu::LayerDescription& layer = model.newLayer();
        layer.addAgentFunction(prey_output_location);
        layer.addAgentFunction(pred_output_location);
        layer.addAgentFunction(grass_output_location);
    }
    {   // Layer #2
        flamegpu::LayerDescription& layer = model.newLayer();
        layer.addAgentFunction(pred_follow_prey);
        layer.addAgentFunction(prey_avoid_pred);
    }
    {   // Layer #3
        flamegpu::LayerDescription& layer = model.newLayer();
        layer.addAgentFunction(prey_flock);
        layer.addAgentFunction(pred_avoid);
    }
    {   // Layer #4
        flamegpu::LayerDescription& layer = model.newLayer();
        layer.addAgentFunction(prey_move);
        layer.addAgentFunction(pred_move);
    }
    {   // Layer #5
        flamegpu::LayerDescription& layer = model.newLayer();
        layer.addAgentFunction(grass_eaten);
        layer.addAgentFunction(prey_eaten);
    }
    {   // Layer #6
        flamegpu::LayerDescription& layer = model.newLayer();
        layer.addAgentFunction(prey_eat_or_starve);
        layer.addAgentFunction(pred_eat_or_starve);
    }
    {   // Layer #7
        flamegpu::LayerDescription& layer = model.newLayer();
        layer.addAgentFunction(pred_reproduction);
        layer.addAgentFunction(prey_reproduction);
        layer.addAgentFunction(grass_growth);
    }

    model.addStepFunction(recordPopulation);
    model.addExitFunction(savePopulationData);
    NVTX_POP();

    /**
     * Create Model Runner
     */
    NVTX_PUSH("CUDAAgentModel creation");
    flamegpu::CUDASimulation cuda_model(model);
    NVTX_POP();

    /**
     * Initialisation
     */
    cuda_model.initialise(argc, argv);

    if (cuda_model.getSimulationConfig().input_file.empty()) {
        printf("Input file was empty!\n");
    }

    // Initialise random number generators
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> floatDist(-1.0f, 1.0f);
    std::uniform_int_distribution<int> predLifeDist(0, 40);
    std::uniform_int_distribution<int> preyLifeDist(0, 50);

    // Load initial population data
    CSVRow initialPops = loadPopulations();

    // Initialise predator agents
    int numPredators = initialPops.predatorPop;
    flamegpu::AgentVector predatorPopulation(model.Agent("predator"), numPredators);
    for (auto predator : predatorPopulation) {
        predator.setVariable<float>("x", floatDist(gen));
        predator.setVariable<float>("y", floatDist(gen));
        predator.setVariable<float>("vx", floatDist(gen));
        predator.setVariable<float>("vy", floatDist(gen));
        predator.setVariable<float>("steer_x", 0.0f);
        predator.setVariable<float>("steer_y", 0.0f);
        predator.setVariable<float>("type", 0.0f);
        predator.setVariable<int>("life", predLifeDist(gen));
    }

    // Initialise prey agents 
    int numPrey = initialPops.preyPop;
    flamegpu::AgentVector preyPopulation(model.Agent("prey"), numPrey);
    for (auto prey : preyPopulation) {
        prey.setVariable<float>("x", floatDist(gen));
        prey.setVariable<float>("y", floatDist(gen));
        prey.setVariable<float>("vx", floatDist(gen));
        prey.setVariable<float>("vy", floatDist(gen));
        prey.setVariable<float>("steer_x", 0.0f);
        prey.setVariable<float>("steer_y", 0.0f);
        prey.setVariable<float>("type", 1.0f);
        prey.setVariable<int>("life", preyLifeDist(gen));
    }

    // Initialise grass agents
    int numGrass = initialPops.grassPop;
    flamegpu::AgentVector grassPopulation(model.Agent("grass"), numGrass);
    for (auto grass : grassPopulation) {
        grass.setVariable<float>("x", floatDist(gen));
        grass.setVariable<float>("y", floatDist(gen));
        grass.setVariable<float>("type", 2.0f);
        grass.setVariable<int>("dead_cycles", 0);
        grass.setVariable<int>("available", 1);

    }

    cuda_model.setPopulationData(grassPopulation);
    cuda_model.setPopulationData(predatorPopulation);
    cuda_model.setPopulationData(preyPopulation);


    /**
     * Execution
     */
    printf("Model initialised, beginning simulation...\n");
    cuda_model.simulate();
    printf("Simulation complete\n");

    return 0;
}

