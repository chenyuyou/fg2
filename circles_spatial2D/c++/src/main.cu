#include "hip/hip_runtime.h"
#include "flamegpu/flamegpu.h"

// Agent Function to output the agents ID and position in to a 2D spatial message list
FLAMEGPU_AGENT_FUNCTION(output_message, flamegpu::MessageNone, flamegpu::MessageSpatial2D) {
    FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getID());
    FLAMEGPU->message_out.setLocation(
        FLAMEGPU->getVariable<float>("x"),
        FLAMEGPU->getVariable<float>("y"));
    return flamegpu::ALIVE;
}

// Agent Function to read the location messages and decide how the agent should move
FLAMEGPU_AGENT_FUNCTION(input_message, flamegpu::MessageSpatial2D, flamegpu::MessageNone) {
    const flamegpu::id_t ID = FLAMEGPU->getID();
    const float REPULSE_FACTOR = FLAMEGPU->environment.getProperty<float>("repulse");
    const float RADIUS = FLAMEGPU->message_in.radius();
    float fx = 0.0;
    float fy = 0.0;
    const float x1 = FLAMEGPU->getVariable<float>("x");
    const float y1 = FLAMEGPU->getVariable<float>("y");
    int count = 0;
    for (const auto &message : FLAMEGPU->message_in(x1, y1)) {
        if (message.getVariable<flamegpu::id_t>("id") != ID) {
            const float x2 = message.getVariable<float>("x");
            const float y2 = message.getVariable<float>("y");
            float x21 = x2 - x1;
            float y21 = y2 - y1;
            const float separation = sqrt(x21*x21 + y21*y21);
            if (separation < RADIUS && separation > 0.0f) {
                float k = sinf((separation / RADIUS)*3.141f*-2)*REPULSE_FACTOR;
                // Normalise without recalculating separation
                x21 /= separation;
                y21 /= separation;
                fx += k * x21;
                fy += k * y21;
                count++;
            }
        }
    }
    fx /= count > 0 ? count : 1;
    fy /= count > 0 ? count : 1;
    FLAMEGPU->setVariable<float>("x", x1 + fx);
    FLAMEGPU->setVariable<float>("y", y1 + fy);
    FLAMEGPU->setVariable<float>("drift", sqrt(fx*fx + fy*fy));
    return flamegpu::ALIVE;
}

FLAMEGPU_INIT_FUNCTION(create_agents) {
    // Fetch the desired agent count and environment width
    const unsigned int AGENT_COUNT = FLAMEGPU->environment.getProperty<unsigned int>("AGENT_COUNT");
    const float ENV_WIDTH = FLAMEGPU->environment.getProperty<float>("ENV_WIDTH");
    // Create agents
    flamegpu::HostAgentAPI t_pop = FLAMEGPU->agent("point");
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        auto t = t_pop.newAgent();
        t.setVariable<float>("x", FLAMEGPU->random.uniform<float>() * ENV_WIDTH);
        t.setVariable<float>("y", FLAMEGPU->random.uniform<float>() * ENV_WIDTH);
    }
}

int main(int argc, const char **argv) {
    // Define some useful constants
    const unsigned int AGENT_COUNT = 16384;
    const float ENV_WIDTH = static_cast<float>(floor(cbrt(AGENT_COUNT)));

    // Define the FLAME GPU model
    flamegpu::ModelDescription model("Circles Tutorial");

    {   // (optional local scope block for cleaner grouping)
        // Define a message of type MessageSpatial2D named location
        flamegpu::MessageSpatial2D::Description message = model.newMessage<flamegpu::MessageSpatial2D>("location");
        // Configure the message list
        message.setMin(0, 0);
        message.setMax(ENV_WIDTH, ENV_WIDTH);
        message.setRadius(1.0f);
        // Add extra variables to the message
        // X Y (Z) are implicit for spatial messages
        message.newVariable<flamegpu::id_t>("id");
    }

    // Define an agent named point
    flamegpu::AgentDescription agent = model.newAgent("point");
    // Assign the agent some variables (ID is implicit to agents, so we don't define it ourselves)
    agent.newVariable<float>("x");
    agent.newVariable<float>("y");
    agent.newVariable<float>("z");
    agent.newVariable<float>("drift", 0.0f);
    // Setup the two agent functions
    flamegpu::AgentFunctionDescription out_fn = agent.newFunction("output_message", output_message);
    out_fn.setMessageOutput("location");
    flamegpu::AgentFunctionDescription in_fn = agent.newFunction("input_message", input_message);
    in_fn.setMessageInput("location");

    {   // (optional local scope block for cleaner grouping)
        // Define environment properties
        flamegpu::EnvironmentDescription env = model.Environment();
        env.newProperty<unsigned int>("AGENT_COUNT", AGENT_COUNT);
        env.newProperty<float>("ENV_WIDTH", ENV_WIDTH);
        env.newProperty<float>("repulse", 0.05f);
    }

    {   // (optional local scope block for cleaner grouping)
        // Dependency specification
        // Message input depends on output
        in_fn.dependsOn(out_fn);
        // Output is the root of our graph
        model.addExecutionRoot(out_fn);
        model.generateLayers();
    }

    model.addInitFunction(create_agents);

    // Specify the desired StepLoggingConfig
    flamegpu::StepLoggingConfig step_log_cfg(model);
    // Log every step
    step_log_cfg.setFrequency(1);
    // Include the mean of the "point" agent population's variable 'drift'
    step_log_cfg.agent("point").logMean<float>("drift");

    // Create the simulation
    flamegpu::CUDASimulation cuda_model(model, argc, argv);

    // Attach the logging config
    cuda_model.setStepLog(step_log_cfg);

// Only compile this block if being built with visualisation support
#ifdef FLAMEGPU_VISUALISATION
    // Create visualisation
    flamegpu::visualiser::ModelVis m_vis = cuda_model.getVisualisation();
    // Set the initial camera location and speed
    const float INIT_CAM = ENV_WIDTH / 2.0f;
    m_vis.setBeginPaused(true);
    m_vis.setInitialCameraTarget(INIT_CAM, INIT_CAM, 0);
    m_vis.setInitialCameraLocation(INIT_CAM, INIT_CAM, ENV_WIDTH);
    m_vis.setCameraSpeed(0.01f);
//    m_vis.setSimulationSpeed(25);
    // Add "point" agents to the visualisation
    flamegpu::visualiser::AgentVis point_agt = m_vis.addAgent("point");
    // Location variables have names "x" and "y" so will be used by default
    point_agt.setModel(flamegpu::visualiser::Stock::Models::ICOSPHERE);
    point_agt.setModelScale(1/10.0f);
    // Mark the environment bounds
    flamegpu::visualiser::LineVis pen = m_vis.newPolylineSketch(1, 1, 1, 0.2f);
    pen.addVertex(0, 0, 0);
    pen.addVertex(0, ENV_WIDTH, 0);
    pen.addVertex(ENV_WIDTH, ENV_WIDTH, 0);
    pen.addVertex(ENV_WIDTH, 0, 0);
    pen.addVertex(0, 0, 0);
    // Open the visualiser window
    m_vis.activate();
#endif

    // Run the simulation
    cuda_model.simulate();

#ifdef FLAMEGPU_VISUALISATION
    // Keep the visualisation window active after the simulation has completed
    m_vis.join();
#endif
}